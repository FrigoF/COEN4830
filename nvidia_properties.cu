// nvidia_properties.cu 
// 
//  Fred J. Frigo
//  07-Oct-2021
//
//
//  See CUDA by Example, J Sanders & E Kandrot, p 33
//
//  To compile:  nvcc nvidia_properties.cu -o nvidia_props
//

#include <stdio.h>
#include <unistd.h>
#include <err.h>
#include <stdint.h>

// you must first call the hipGetDeviceProperties() function, then pass 
// the devProp structure returned to this function:
int getSPcores(hipDeviceProp_t devProp)
{  
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
     case 2: // Fermi
      if (devProp.minor == 1) cores = mp * 48;
      else cores = mp * 32;
      break;
     case 3: // Kepler
      cores = mp * 192;
      break;
     case 5: // Maxwell
      cores = mp * 128;
      break;
     case 6: // Pascal
      if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
      else if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 7: // Volta and Turing
      if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 8: // Ampere
      if (devProp.minor == 0) cores = mp * 64;
      else if (devProp.minor == 6) cores = mp * 128;
      else printf("Unknown device type\n");
      break;
     default:
      printf("Unknown device type\n"); 
      break;
      }
    return cores;
}

int main( void ) 
{
    hipDeviceProp_t prop;
    int count;

    hipGetDeviceCount( &count );

    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( " --- General Information for device %d ---\n", i );
        printf( "Name: %s\n", prop.name );
        printf( "Total cores = %d\n", getSPcores(prop));
        printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate: %d\n", prop.clockRate );
        printf( "Device copy overlap: " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );
        printf( "Kernel execition timeout : " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );
        printf( " --- Memory Information for device %d ---\n", i );
        printf( "Total global mem: %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem: %ld\n", prop.totalConstMem );
        printf( "Max mem pitch: %ld\n", prop.memPitch );
        printf( "Texture Alignment: %ld\n", prop.textureAlignment );
        printf( " --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count: %d\n", prop.multiProcessorCount );
        printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp: %d\n", prop.regsPerBlock );
        printf( "Threads in warp: %d\n", prop.warpSize );
        printf( "Max threads per block: %d\n", prop.maxThreadsPerBlock );
        printf( "Max thread dimensions: (%d, %d, %d)\n",
            prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
        printf( "Max grid dimensions: (%d, %d, %d)\n",
            prop.maxGridSize[0], prop.maxGridSize[1],prop.maxGridSize[2] );
        printf( "\n" );
    }
    return(0);
}
