#include "hip/hip_runtime.h"
// mandelbrot.cu 
// 
//  Fred J. Frigo
//  28-Aug-2021
//
//  See section B19.4:
//  https://docs.nvidia.com/cuda/archive/9.1/pdf/CUDA_C_Programming_Guide.pdf`
//
//  See also;
//  http://selkie.macalester.edu/csinparallel/modules/CUDAArchitecture/build/html/1-Mandelbrot/Mandelbrot.html
//
//  To compile:  nvcc mandelbrot.cu -o mandelbrot
//

#include <stdio.h>

__device__ uint32_t mandel_double(double cr, double ci, int max_iter) {
    double zr = 0;
    double zi = 0;
    double zrsqr = 0;
    double zisqr = 0;

    uint32_t i;

    for (i = 0; i < max_iter; i++){
		zi = zr * zi;
		zi += zi;
		zi += ci;
		zr = zrsqr - zisqr + cr;
		zrsqr = zr * zr;
		zisqr = zi * zi;
		
    //the fewer iterations it takes to diverge, the farther from the set
		if (zrsqr + zisqr > 4.0) break;
    }
	
    return i;
}


__global__ void mandel_kernel(uint32_t *counts, double xmin, double ymin,
            double step, int max_iter, int dim, uint32_t *colors) {
    int pix_per_thread = dim * dim / (gridDim.x * blockDim.x);
    int tId = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = pix_per_thread * tId;
    for (int i = offset; i < offset + pix_per_thread; i++){
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = colors[mandel_double(cr, ci, max_iter)];
    }
    if (gridDim.x * blockDim.x * pix_per_thread < dim * dim
            && tId < (dim * dim) - (blockDim.x * gridDim.x)){
        int i = blockDim.x * gridDim.x * pix_per_thread + tId;
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = colors[mandel_double(cr, ci, max_iter)];
    }
    
}

int main()
{ 
   
   mandel_kernel<<<n, m>>>(dev_counts, xmin , ymin, step, max_iter, dim, colors);
   hipDeviceSynchronize();
   printf(“CUDA Mandelbrot\n”); 
   return 0;
}
